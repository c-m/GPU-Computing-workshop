#include "hip/hip_runtime.h"
#include "Utils.h"

// ===============
// The CUDA kernel
// ===============

__global__ void helloworldKernel(const int nGridSize)
{
	int gIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if(gIdx >= nGridSize)
	{
		return;
	}

	printf("Hello, world! from thread(tIdx = %d, bIdx = %d, gIdx = %d)" NEW_LINE, threadIdx.x, blockIdx.x, gIdx);
}

// =================
// Run the CUDA grid
// =================

void runDeviceGrid(const int nBlocks, const int nThreads, const int nGridSize)
{
	// Launch the grid asynchronously
	helloworldKernel<<<nBlocks, nThreads>>>(nGridSize);

	// Wait for the grid to finish
	SAFE_CUDA_CALL(hipDeviceSynchronize());
}

// =======================
// Application entry point
// =======================

int _01_Hello_World(int argCount, char ** argValues)
{
	int vGridConf[3];
	const char * vErrMessages[2] =	{"Error: The number of threads must be greater than 0.",
									 "Error: The grid size must be greater than 0."};

	// Extract and validate the number of blocks and threads to launch
	validateArguments(argCount, 2, argValues, vGridConf, vErrMessages);

	printf("Starting application (B: %d, T: %d, G: %d):" NEW_LINE, vGridConf[0], vGridConf[1], vGridConf[2]);

	// Launch the CUDA grid
	runDeviceGrid(vGridConf[0], vGridConf[1], vGridConf[2]);

	printf("The application has finished." NEW_LINE);

	WAIT_AND_EXIT(0);
}